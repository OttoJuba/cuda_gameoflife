#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>
#include <sys/time.h>
#include <unistd.h>


#define BLOCK_SIDE 16

typedef unsigned char ubyte;
int writeArr[64][64];

__device__ ubyte getat(ubyte* pboard, int nrows, int ncols, int x, int y)
{
	if (x >= 0 && x < ncols && y >= 0 && y < nrows)
		return pboard[x * ncols + y];
	return 0x0;
}

__device__ int numneighbors(int x, int y, ubyte* pboard, int nrows, int ncols)
{
	int num = 0;

	num += (getat(pboard, nrows, ncols, x-1, y));

	num += (getat(pboard, nrows, ncols, x+1, y));
	
	num += (getat(pboard, nrows, ncols, x, y-1));
	
	num += (getat(pboard, nrows, ncols, x, y+1));
	
	num += (getat(pboard, nrows, ncols, x-1, y-1));
	
	num += (getat(pboard, nrows, ncols, x-1, y+1));
	
	num += (getat(pboard, nrows, ncols, x+1, y-1));
	
	num += (getat(pboard, nrows, ncols, x+1, y+1));
	
	return num;
}

__global__ void simstep(int nrows, int ncols, ubyte* pCurrBoard, ubyte* pNewBoard)
{
	int x = blockIdx.x * BLOCK_SIDE + threadIdx.x;
	int y = blockIdx.y * BLOCK_SIDE + threadIdx.y;

	int indx = x * ncols + y;

	pNewBoard[indx] = pCurrBoard[indx];

	int neighbors = numneighbors(x, y, pCurrBoard, nrows, ncols);

	// Apply game rules:
	// "Any live cell with fewer than two live neighbours dies, 
	// as if caused by under-population." [WIK11]
	if (neighbors < 2)
		pNewBoard[indx] = 0x0;

	// "Any live cell with two or three live neighbours lives on 
	// to the next generation." [WIK11]
	// (do nothing)

	// "Any live cell with more than three live neighbours dies, 
	// as if by overcrowding." [WIK11]
	if (neighbors > 3)
		pNewBoard[indx] = 0x0;

	// "Any dead cell with exactly three live neighbours becomes 
	// a live cell, as if by reproduction." [WIK11]
	if (neighbors == 3 && !pCurrBoard[indx])
		pNewBoard[indx] = 0x1;
}

void randomizeBoard(ubyte* pboard, int nrows, int ncols, float probability)
{
	for (int x = 0; x < ncols/2; x++)
	{
		for (int y = 0; y < nrows/2; y++)
		{
			float rnd = rand() / (float)RAND_MAX;
			pboard[x * ncols + y] = (rnd >= probability)? 0x1 : 0x0;
		}
	}
}

void printBoard(const char* msg, ubyte* pboard, int nrows, int ncols)
{
	printf("%s\n", msg);

	for (int x = 0; x < ncols; x++)
	{
		for (int y = 0; y < nrows; y++)
		{
			printf("%c ", pboard[x * ncols + y]? 'o' : ' ');
		}
		printf("\n");
	}

}

void writeBoard(ubyte* pboard, int boardH, int boardW, int board[64][64])
{

	for (int x = 0; x < boardH; x++)
	{
		for (int y = 0; y < boardW; y++)
		{
			board[x][y] = (int)(pboard[x * boardW + y]? '0' : '1');		
			//printf("%c ", pboard[x * ncols + y]? '0' : '1');
		}
	}

}

int main(int argc, char* argv[])
{
	FILE *fp;
	fp = fopen("/home/otto/Documents/graphics/game-of-life/data.txt","r+");

	int boardW = 64;
	int boardH = 64;

	int ngenerations = 1000000;
	if (argc > 1)
	{
		ngenerations = atoi(argv[1]);
	}

	printf("Running %d generations\n", ngenerations);

	srand(time(0));

	ubyte* pboard = (ubyte *)malloc(boardW * boardH * sizeof(ubyte));
	randomizeBoard(pboard, boardH, boardW, 0.7f);
	printBoard("Initial Board:", pboard, boardH, boardW);

	ubyte* pDevBoard0;
	hipMalloc((void **)&pDevBoard0, boardW * boardH * sizeof(ubyte));
	hipMemcpy(pDevBoard0, pboard, boardH * boardW * sizeof(ubyte), hipMemcpyHostToDevice);

	ubyte* pDevBoard1;
	hipMalloc((void **)&pDevBoard1, boardW * boardH * sizeof(ubyte));
	hipMemset(pDevBoard1, 0x0, boardH * boardW * sizeof(ubyte));

	dim3 blocksize(BLOCK_SIDE, BLOCK_SIDE);
	dim3 gridsize(boardW / BLOCK_SIDE, boardH / BLOCK_SIDE);

	struct timeval ti;
	gettimeofday(&ti, NULL);

	ubyte* pcurr;
	ubyte* pnext;
	for (int gen = 0; gen < ngenerations; gen++)
	{
		if ((gen % 2) == 0)
		{
			pcurr = pDevBoard0;
			pnext = pDevBoard1;
		}
		else
		{
			pcurr = pDevBoard1;

			pnext = pDevBoard0;
		}
		hipMemcpy(pboard, pnext, boardH * boardW * sizeof(ubyte), hipMemcpyDeviceToHost);

		for (int i = 0; i < 24; i++) printf("\n");
		printBoard(" ", pboard, boardH, boardW);
		usleep(70000);
		simstep<<<gridsize, blocksize>>>(boardH, boardW, pcurr, pnext);

		writeBoard(pboard, boardH, boardW, writeArr);
		fwrite(writeArr,1,sizeof(writeArr),fp);


#ifdef PRINT_BOARDS
		hipMemcpy(pboard, pnext, boardH * boardW * sizeof(ubyte), hipMemcpyDeviceToHost);
		for (int i = 0; i < 10; i++) printf("\n");
		printBoard(" ", pboard, boardH, boardW);
		//usleep(250000);
#endif



	}


	struct timeval tf;
	gettimeofday(&tf, NULL);
	double t = ((tf.tv_sec - ti.tv_sec) * 1000.0) + ((tf.tv_usec - ti.tv_usec) / 1000.0);


	hipMemcpy(pboard, pcurr, boardW * boardH * sizeof(ubyte), hipMemcpyDeviceToHost);



	printBoard("Resulting Board:", pboard, boardH, boardW);

	hipFree(pDevBoard0);
	hipFree(pDevBoard1);
	free(pboard);
	fclose(fp);

	printf("%d generations in %f milliseconds\n", ngenerations, t);

	return 0;
}

